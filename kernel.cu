#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <iostream>

// Kernel to compute force matrix
// Kernel to compute force matrix
__global__ void computeForces(double* forceX, double* forceY, double* xPos, double* yPos,
    int N, int A, int B, double* sigma, const double epsilon)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < N && j < N && i != j)
    {
        double dx = xPos[i] - xPos[j];
        double dy = yPos[i] - yPos[j];
        double sep = sqrt(dx * dx + dy * dy);

        // Avoid division by zero and self-interaction
        if (sep > 1e-9) {  // Use a small threshold to avoid zero division
            double invr7 = 1.0 / pow(sep, 7);
            double invr13 = 1.0 / pow(sep, 13);

            // Calculate force
            double force = 4 * epsilon * ((A * pow(sigma[i], 6) * invr7) - (B * pow(sigma[i], 12) * invr13));
            forceX[i * N + j] = -(dx / sep) * force;
            forceY[i * N + j] = -(dy / sep) * force;
        }
        else {
            forceX[i * N + j] = 0.0;
            forceY[i * N + j] = 0.0;
        }
    }
}

// Kernel to aggregate accelerations
// Kernel to aggregate accelerations
__global__ void aggregateAccelerations(double* forceX, double* forceY, double* accX, double* accY, const double* masses, int N)
{
    int rowIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (rowIdx < N)
    {
        double sumX = 0;
        double sumY = 0;
        for (int col = 0; col < N; ++col)
        {
            sumX += forceX[rowIdx * N + col];
            sumY += forceY[rowIdx * N + col];
        }

        // Avoid NaN in acceleration calculation
        if (masses[rowIdx] != 0.0) {
            accX[rowIdx] = sumX / masses[rowIdx];
            accY[rowIdx] = sumY / masses[rowIdx];
        }
        else {
            accX[rowIdx] = 0.0;
            accY[rowIdx] = 0.0;
        }
    }
}



__global__ void integratePositions(int count, double* dev_xPosMatrix, double* dev_yPosMatrix, double* xPos, double* yPos, double* xVel, double* yVel, double* accX, double* accY, int N, double timeStep, double* radii, int boxwidth) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        if (count % 100 == 0) {
            int row = count / 100;
            dev_xPosMatrix[row * N + i] = xPos[i];
            dev_yPosMatrix[row * N + i] = yPos[i];
        }

        xPos[i] += xVel[i] * timeStep + 0.5 * accX[i] * timeStep * timeStep;
        yPos[i] += yVel[i] * timeStep + 0.5 * accY[i] * timeStep * timeStep;

        // Handle boundary conditions after position update
        if (xPos[i] - radii[i] <= 0 || xPos[i] + radii[i] >= boxwidth) {
            xVel[i] = -xVel[i];
        }
        if (yPos[i] - radii[i] <= 0 || yPos[i] + radii[i] >= boxwidth) {
            yVel[i] = -yVel[i];
        }
    }
}


__global__ void integrateVelocities(double* xVel, double* yVel, double* oldAccX, double* oldAccY, double* newAccX, double* newAccY, int N, double timeStep) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
    {
        xVel[i] += 0.5 * (newAccX[i] + oldAccX[i]) * timeStep;
        yVel[i] += 0.5 * (newAccY[i] + oldAccY[i]) * timeStep;

        oldAccX[i] = newAccX[i];
    }
}



// Function to compute accelerations
hipError_t computeAccelerations(double* dev_xPos, double* dev_yPos, double* dev_masses, double* dev_accX, double* dev_accY, double* dev_sigma,
    const unsigned int N, const double A, const double B, const double epsilon, const double timeStep)
{
    double* dev_forceX;
    double* dev_forceY;
    hipMalloc((void**)&dev_forceX, N * sizeof(double));
    hipMalloc((void**)&dev_forceY, N * sizeof(double));

    // Define the grid and block dimensions
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Launch the kernel on the GPU
    computeForces <<<blocksPerGrid, threadsPerBlock>>> (dev_forceX, dev_forceY, dev_xPos, dev_yPos,
        N, A, B, dev_sigma, epsilon);
    hipError_t cudaStatus = hipDeviceSynchronize();
    // Aggregate accelerations
    aggregateAccelerations <<<blocksPerGrid, threadsPerBlock.x >>> (dev_forceX, dev_forceY, dev_accX, dev_accY, dev_masses, N);
    cudaStatus = hipDeviceSynchronize();

    // Free GPU buffers
    hipFree(dev_forceX);
    hipFree(dev_forceY);

    return cudaStatus;
}


#include <stdio.h>

// Function to write a matrix to a CSV file
void writeMatrixToFile(double* matrix, int rows, int cols, const char* filename) {
    FILE* file = fopen(filename, "w");
    if (file == NULL) {
        fprintf(stderr, "Error opening file %s for writing.\n", filename);
        return;
    }
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            fprintf(file, "%f", matrix[i * cols + j]);
            if (j < cols - 1) fprintf(file, ","); // No trailing comma at the end of the row
        }
        fprintf(file, "\n"); // New line at the end of each row
    }
    fclose(file);
}


void printMatrix(double* matrix, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%f ", matrix[i*cols + j]);
        }
        printf("\n");
    }
}





int main()
{
    const int N = 10;
    const double epsilon = 0.1;
    const double A = 1;
    const double B = 1;
    const double timeStep = 1E-4;
    const int runTime = 10;
    const int iterations = runTime / timeStep;
    const int boxwidth = 25;

    // Allocate memory
    double* xPos = (double*)malloc(N * sizeof(double));
    double* yPos = (double*)malloc(N * sizeof(double));
    double* xVel = (double*)malloc(N * sizeof(double));
    double* yVel = (double*)malloc(N * sizeof(double));
    double* masses = (double*)malloc(N * sizeof(double));
    double* sigma = (double*)malloc(N * sizeof(double));
    double* radii = (double*)malloc(N * sizeof(double));
    double* xPositionMatrix = (double*)malloc(runTime*100 * N * sizeof(double));
    double* yPositionMatrix = (double*)malloc(runTime*100 * N * sizeof(double));

    // Initialize positions, velocities, etc.
    srand(time(NULL));
    for (int i = 0; i < N; i++) {
        masses[i] = 1;
        xPos[i] = (double)rand() / (double)(RAND_MAX / 20);
        yPos[i] = (double)rand() / (double)(RAND_MAX / 20);
        xVel[i] = (double)rand() / (double)(RAND_MAX / 4);
        yVel[i] = (double)rand() / (double)(RAND_MAX / 4);
        radii[i] = 0.3;
        sigma[i] = 0.3/pow(2,1/6);
    }
    masses[N / 2] = 1; // Brownian particle
    radii[N / 2] = 0.7;

    // Allocate device memory
    double* dev_xPos, * dev_yPos, * dev_xVel, * dev_yVel, * dev_accX, * dev_accY;
    double* dev_newaccX, * dev_newaccY, * dev_sigma, * dev_masses, * dev_radii;
    double* dev_xmat, * dev_ymat;

    hipMalloc((void**)&dev_xPos, N * sizeof(double));
    hipMalloc((void**)&dev_yPos, N * sizeof(double));
    hipMalloc((void**)&dev_xVel, N * sizeof(double));
    hipMalloc((void**)&dev_yVel, N * sizeof(double));
    hipMalloc((void**)&dev_accX, N * sizeof(double));
    hipMalloc((void**)&dev_accY, N * sizeof(double));
    hipMalloc((void**)&dev_newaccX, N * sizeof(double));
    hipMalloc((void**)&dev_newaccY, N * sizeof(double));
    hipMalloc((void**)&dev_sigma, N * sizeof(double));
    hipMalloc((void**)&dev_masses, N * sizeof(double));
    hipMalloc((void**)&dev_radii, N * sizeof(double));
    hipMalloc((void**)&dev_xmat, runTime*100 * N * sizeof(double));
    hipMalloc((void**)&dev_ymat, runTime*100 * N * sizeof(double));

    // Copy data to device
    hipMemcpy(dev_xPos, xPos, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_yPos, yPos, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_xVel, xVel, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_yVel, yVel, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_sigma, sigma, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_masses, masses, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_radii, radii, N * sizeof(double), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x);

    // Compute initial accelerations
    computeAccelerations(dev_xPos, dev_yPos, dev_masses, dev_accX, dev_accY, dev_sigma, N, A, B, epsilon, timeStep);


    // Main loop
    for (int count = 0; count < iterations; count++) {
        integratePositions << <blocksPerGrid, threadsPerBlock >> > (count, dev_xmat, dev_ymat, dev_xPos, dev_yPos, 
            dev_xVel, dev_yVel, dev_accX, dev_accY, N, timeStep, dev_radii, boxwidth);
        hipDeviceSynchronize();

        // Compute new accelerations after positions are updated
        computeAccelerations(dev_xPos, dev_yPos, dev_masses, dev_newaccX, dev_newaccY, dev_sigma, N, A, B, epsilon, timeStep);
        hipDeviceSynchronize();

        // Update velocities using old and new accelerations
        integrateVelocities << <blocksPerGrid, threadsPerBlock >> > (dev_xVel, dev_yVel, dev_accX, dev_accY, 
            dev_newaccX, dev_newaccY, N, timeStep);
        hipDeviceSynchronize();

    }


    // Copy results back to host
    hipMemcpy(xPositionMatrix, dev_xmat, runTime*100 * N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(yPositionMatrix, dev_ymat, runTime*100 * N * sizeof(double), hipMemcpyDeviceToHost);

    writeMatrixToFile(xPositionMatrix, runTime * 100, N, "xPositionMatrix.csv");
    writeMatrixToFile(yPositionMatrix, runTime * 100, N, "yPositionMatrix.csv");



    // Free memory
    free(xPos);
    free(yPos);
    free(xVel);
    free(yVel);
    free(masses);
    free(sigma);
    free(radii);
    free(xPositionMatrix);
    free(yPositionMatrix);

    hipFree(dev_xPos);
    hipFree(dev_yPos);
    hipFree(dev_xVel);
    hipFree(dev_yVel);
    hipFree(dev_accX);
    hipFree(dev_accY);
    hipFree(dev_newaccX);
    hipFree(dev_newaccY);
    hipFree(dev_sigma);
    hipFree(dev_masses);
    hipFree(dev_radii);
    hipFree(dev_xmat);
    hipFree(dev_ymat);

    return 0;
}
